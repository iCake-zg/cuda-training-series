
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n",blockIdx.x, threadIdx.x);
}

int main(){
  // 2 blocks &  2 threads
  hello<<<2,2>>>();
  hipDeviceSynchronize();
}

